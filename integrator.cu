#include "hip/hip_runtime.h"
#include "integrator.cuh"

Integrator::Integrator() : CPUintegral(0.0) {

    hipMalloc(&pdGPUintegral, sizeof(float));
    hipMemcpy(pdGPUintegral, &CPUintegral, sizeof(float), hipMemcpyHostToDevice);
} 

Integrator::~Integrator() {
    hipFree(pdGPUintegral);
}

float Integrator::CPUintegrator(const thrust::host_vector<float> &vecx,const thrust::host_vector<float> &vecy) {

    for(int i = 0; i < vecx.size() - 1; i++) {
        // std::cout << "current integral value is " << CPUintegral << " value to be added " << ((vecy[i] + vecy[i+1]) * (vecx[i+1] - vecx[i]) ) / 2 << std::endl;
        CPUintegral += ((vecy[i] + vecy[i+1]) * (vecx[i+1] - vecx[i]) ) / 2 ; // trapezoidal integration
    }

    return CPUintegral;
}
////////////////////////////////////////////////////////////////////////
// GPU implementation

__global__ void dGPUintegrator(float *dvecx, 
                               float *dvecy,
                               float *integral,
                               int *maxindex) {

    int index = threadIdx.x + blockIdx.x * blockDim.x;

    if(index < *maxindex - 1) {
        
        atomicAdd(integral, ((dvecy[index] + dvecy[index+1]) * (dvecx[index+1] - dvecx[index]) ) / 2);
    }
}

float Integrator::GPUintegrator(const thrust::host_vector<float> &vecx,
                                 const thrust::host_vector<float> &vecy)
{
    // allocate GPU memory
    thrust::device_vector<float> dvecx = vecx;
    thrust::device_vector<float> dvecy = vecy;

    // get device pointer
    thrust::device_ptr<float> pdvecx = dvecx.data();
    thrust::device_ptr<float> pdvecy = dvecy.data();

    int maxindex = vecx.size();
    int *pmaxindex;
    hipMalloc(&pmaxindex, sizeof(int));
    hipMemcpy(pmaxindex, &maxindex, sizeof(int), hipMemcpyHostToDevice );

    // choose number of threads and blocks
    dim3 threadnum = 256;
    dim3 blocknum =  vecx.size() / threadnum.x + 1; 

    dGPUintegrator<<<blocknum, threadnum>>>(pdvecx.get(), pdvecy.get(), this->pdGPUintegral, pmaxindex);

    hipMemcpy(&GPUintegral, this->pdGPUintegral, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(pmaxindex);

    return this->GPUintegral;
}